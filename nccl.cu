#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>

#include <nccl.h>
__global__ void show(int *in,int i){
    in[threadIdx.x] = i;
    printf("%d\n",in[threadIdx.x]);
}

__global__ void show_al(int *in){
    //printf("%d\n",in[threadIdx.x]);
}

int main(int argc, char* argv[]) {

  /*Get current amounts number of GPU*/
  int nGPUs = 0;
  hipGetDeviceCount(&nGPUs);
  printf("nGPUs = %d\n",nGPUs);

  /*List GPU Device*/
  int *DeviceList;  
  DeviceList = (int *)malloc( nGPUs * sizeof(int));
  for (int i = 0; i < nGPUs; ++i){
      DeviceList[i] = i;
  }
  
  /*NCCL Init*/
  ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*nGPUs);  
  hipStream_t* s = (hipStream_t*)malloc(sizeof(hipStream_t)*nGPUs);
  ncclCommInitAll(comms, nGPUs, DeviceList);
  /*Get GPU status*/
  printf("# Using devices\n");
  for (int g = 0; g < nGPUs; g++) {
      int cudaDev;
      int rank;
      hipDeviceProp_t prop;
      ncclCommCuDevice(comms[g], &cudaDev);
      ncclCommUserRank(comms[g], &rank);

      hipGetDeviceProperties(&prop, cudaDev);
      printf("#   Rank %2d uses device %2d [0x%02x] %s\n", rank, cudaDev, prop.pciBusID, prop.name);
  }
  printf("\n");

  /*Malloc the data*/
  int data_size = 2000000000 ;
  int* data;
  data = (int*)malloc(data_size * sizeof(int));
  int **d_data;
  d_data = (int**)malloc(nGPUs * sizeof(int*));
  int **d_data_al;
  d_data_al = (int**)malloc(nGPUs * sizeof(int*));


  for(int i = 0; i < data_size; i++){
      data[i] = i;
  }


  for(int g = 0; g < nGPUs; g++) {
      char busid[32] = {0};
      hipDeviceGetPCIBusId(busid, 32, DeviceList[g]);
      printf("# Rank %d using device %d [%s]\n", g, DeviceList[g], busid);

      hipSetDevice(DeviceList[g]);
      hipStreamCreate(&s[g]);
      hipMalloc(&d_data[g], data_size * sizeof(int));
      hipMalloc(&d_data_al[g], data_size * sizeof(int));

      if(g == 0) {
          hipMemcpy(d_data[g], data, data_size * sizeof(int),hipMemcpyHostToDevice);
      }
  }

  

  // GPU Bcast
  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      ncclBcast(d_data[i], data_size, ncclInt, 0, comms[i], s[i]);
  }


  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      printf("This is device %d \n",i);
      //show<<<1,8>>>(d_data[i],i);
      hipDeviceSynchronize();
  }
  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      hipStreamSynchronize(s[i]);
  }

  printf("Bcast Done!\n");

  // GPU Allgather
  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      ncclAllGather(d_data[i], 1, ncclInt, d_data_al[i], comms[i], s[i]);
  }
  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      printf("This is device %d \n",i);
      //show_al<<<1,8>>>(d_data_al[i]);
      hipDeviceSynchronize();
  }
  for (int i = 0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      hipStreamSynchronize(s[i]);
  }

  printf("Allgather Done!\n");


  for(int i=0; i < nGPUs; ++i) {
      hipSetDevice(DeviceList[i]);
      hipStreamDestroy(s[i]);
  }

  printf("StreamDestroy Done!\n");

  free(s);
  hipFree(d_data);
  hipFree(d_data_al);
}

